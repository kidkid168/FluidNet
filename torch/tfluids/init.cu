#include "hip/hip_runtime.h"
// Copyright 2016 Google Inc, NYU.
// 
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// 
//     http://www.apache.org/licenses/LICENSE-2.0
// 
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <assert.h>

#include <algorithm>

#include <TH.h>
#include <luaT.h>
#include "generic/tfluids.cu.h"
#include "generic/stack_trace.cc"

// This type is common to both float and double implementations and so has
// to be defined outside tfluids.cc.
typedef struct Int3 {
  int32_t x;
  int32_t y;
  int32_t z;
} Int3;

inline int32_t IX(const int32_t i, const int32_t j, const int32_t k,
                  const Int3& dims) {
#if defined(DEBUG)
  assert(i >= 0 && i < dims.x);
  assert(j >= 0 && j < dims.y);
  assert(k >= 0 && k < dims.z);
#endif
  return i + j * dims.x + k * dims.x * dims.y;
}

inline int32_t ClampInt32(const int32_t x, const int32_t low,
                          const int32_t high) {
  return std::max<int32_t>(std::min<int32_t>(x, high), low);
}

#define torch_(NAME) TH_CONCAT_3(torch_, Real, NAME)
#define torch_Tensor TH_CONCAT_STRING_3(torch., Real, Tensor)
#define tfluids_(NAME) TH_CONCAT_3(tfluids_, Real, NAME)

// Note: instead of calling THGenerateFloatTypes.h, we're going to hack into
// the torch build system a little bit. This makes the tfluids library
// compatible with the blaze build system (for reasons that aren't interesting,
// but are very annoying).
#define TH_GENERIC_FILE

#define real float
#define accreal double
#define Real Float
#define THInf FLT_MAX
#define TH_REAL_IS_FLOAT
#include "generic/tfluids.cc"
#undef accreal
#undef real
#undef Real
#undef THInf
#undef TH_REAL_IS_FLOAT

#define real double
#define accreal double
#define Real Double
#define THInf DBL_MAX
#define TH_REAL_IS_DOUBLE
#include "generic/tfluids.cc"
#undef accreal
#undef real
#undef Real
#undef THInf
#undef TH_REAL_IS_DOUBLE

#undef TH_GENERIC_FILE

LUA_EXTERNC DLL_EXPORT int luaopen_libtfluids(lua_State *L) {
  tfluids_FloatMain_init(L);
  tfluids_DoubleMain_init(L);
  tfluids_CudaMain_init(L);

  lua_newtable(L);
  lua_pushvalue(L, -1);
  lua_setglobal(L, "tfluids");

  lua_newtable(L);
  luaT_setfuncs(L, tfluids_DoubleMain__, 0);
  lua_setfield(L, -2, "double");

  lua_newtable(L);
  luaT_setfuncs(L, tfluids_FloatMain__, 0);
  lua_setfield(L, -2, "float");

  lua_newtable(L);
  luaT_setfuncs(L, tfluids_CudaMain_getMethodsTable(), 0);
  lua_setfield(L, -2, "cuda");

  return 1;
}
